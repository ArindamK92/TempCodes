#include "hip/hip_runtime.h"
/*extern "C" {} *///it will instruct the compiler to expect C linkage for your C functions, not C++ linkage.
#include <thrust/find.h>
#include <thrust/device_vector.h>
#include <thrust/count.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/device_free.h>
#include <stdio.h>
//#include "all_structures.h"
#include "all_structure_cuda.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>


#include<vector>
#include <chrono> 


#define THREADS_PER_BLOCK 1024 //we can change it

using namespace std;
using namespace std::chrono;

__global__ void initialize(int nodes, int src, RT_Vertex* SSSP, int* stencil)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < nodes)
	{
		if (index == src) { SSSP[index].Root = -1; } //try to avoid
		else { SSSP[index].Root = index; }
		SSSP[index].Dist = 0.0;
		stencil[index] = index;
	}
}

//__global__ void create_tree(Colwt2* cuda_adjlist_full_X, int start, RT_Vertex* SSSP, int src, int* d_affectedPointer, int numberofCudaThread)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//	int number_CudaThread = numberofCudaThread;
//	int flag = 0;
//
//	if (index < number_CudaThread)
//	{
//		/*printf("source: %d", src);*/
//		int y = cuda_adjlist_full_X[index + start].col;
//		/*printf("y: %d", y);*/
//		double mywt = cuda_adjlist_full_X[index + start].wt;
//		if (mywt == -1) { flag = 1; }//invalid edge
//		if (SSSP[y].Root == -1) { flag = 1; }
//		if (flag == 0)
//		{
//			SSSP[y].Parent = src; //mark the parent
//			SSSP[y].EDGwt = mywt; //mark the edgewt
//			SSSP[y].Root = SSSP[src].Root;
//			SSSP[y].Dist = SSSP[src].Dist + mywt;
//			d_affectedPointer[y] = 1;
//		}
//		/*printf("end if***");*/
//
//	}
//
//}

__global__ void create_tree2(Colwt2* cuda_adjlist_full_X, int* d_colStartPtr_X, RT_Vertex* SSSP, int* d_affectedPointer, int* change_d, int nodes)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int flag = 0;
	
	if (index < nodes && d_affectedPointer[index] == 1)
	{
		/*printf("index: %d", index);
		printf("affected ptr value: %d", d_affectedPointer[index]);*/
		int adjNodestartIndex = d_colStartPtr_X[index];
		/*printf("y: %d", y);*/
		for (int k = 0; k < d_colStartPtr_X[index + 1] - d_colStartPtr_X[index]; k++)
		{
			flag = 0;
			int y = cuda_adjlist_full_X[adjNodestartIndex + k].col;
			double mywt = cuda_adjlist_full_X[adjNodestartIndex + k].wt;
			/*printf("y: %d", y);*/
			/*printf("SSSP[y].Root:%d,SSSP[y].Parent:%d, SSSP[y].Root:%d,SSSP[y].Dist:%f", SSSP[y].Root, SSSP[y].Parent, SSSP[y].Root, SSSP[y].Dist);*/
			if (mywt == -1) {
				/*printf("check 1.1");*/
				flag = 1; }//invalid edge
			if (SSSP[y].Root == -1) { /*printf("check 1.2");*/ flag = 1; }
			if (SSSP[y].Root != y) { /*printf("check 1.3");*/ flag = 1; }
			if (flag == 0)
			{
				SSSP[y].Parent = index; //mark the parent
				SSSP[y].EDGwt = mywt; //mark the edgewt
				SSSP[y].Root = SSSP[index].Root;
				SSSP[y].Dist = SSSP[index].Dist + mywt;
				/*printf("mywt: %f", mywt);*/
				d_affectedPointer[y] = 1;
			}
		}
		d_affectedPointer[index] = 0;
		change_d[0] = 1;
		/*printf("end if***");*/

	}

}

struct is_affected
{
	__host__ __device__
		bool operator()(const int x)
	{
		return (x == 1);
	}
};

__global__ void initializeUpdatedDist(double* d_UpdatedDist, RT_Vertex* SSSP, int X_size)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < X_size)
	{
		d_UpdatedDist[index] = SSSP[index].Dist;
	}
}

__global__ void initializeEdgedone(int* Edgedone, int totalChange)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < totalChange)
	{
		Edgedone[index] = -1;
	}
}

__global__ void insertDeleteEdge(xEdge_cuda* allChange_cuda, int* Edgedone, RT_Vertex* SSSP, int numS, int X_size, int* d_colStartPtr_X, Colwt2* cuda_adjlist_full_X, double* d_UpdatedDist, double inf, Colwt2* cuda_adjlist_full_R, int* colStartPtr_R)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < numS)
	{
		int node_1 = allChange_cuda[index].node1;
		int node_2 = allChange_cuda[index].node2;
		double edge_weight = allChange_cuda[index].edge_wt;

		if (node_1 > X_size || allChange_cuda[index].node2 > X_size)
		{
			Edgedone[index] = 0; //mark to not add
		}

		if (SSSP[node_1].Root != SSSP[node_2].Root)
		{
			Edgedone[index] = 0; //mark to not add
		}

		if (allChange_cuda[index].inst == 1)  //check x
		{
			//Check if edge exists--then dont insert 
			for (int k = 0; k < d_colStartPtr_X[node_1 + 1] - d_colStartPtr_X[node_1]; k++)
			{
				int myn = cuda_adjlist_full_X[d_colStartPtr_X[node_1] + k].col;
				double mywt = cuda_adjlist_full_X[d_colStartPtr_X[node_1] + k].wt; //check. added recently 01-15-20
				//****need check
				if (myn == node_2 && mywt <= edge_weight && mywt != -1)
				{
					Edgedone[index] = 0;
					break;
				}

			}//end of for
		}

		if (allChange_cuda[index].inst == 1 && Edgedone[index] != 0)
		{
			//We check the distances based on updateddist, to cull some insertion edges
			//In case of conflicts, actual distance remains correct

				//Default is remainder edge
			Edgedone[index] = 2;
			//Check twice once for  n1->n2 and once for n2->n1
			for (int yy = 0; yy < 2; yy++)
			{
				int node1, node2;
				if (yy == 0)
				{
					node1 = node_1;
					node2 = node_2;
				}
				else
				{
					node1 = node_2;
					node2 = node_1;
				}

				//  printf("%d:%f:::%d::%f:::%f \n", node1, UpdatedDist[node1],node2, UpdatedDist[node2], mye.edge_wt);
		  //Check whether node1 is relaxed
				if (d_UpdatedDist[node2] > d_UpdatedDist[node1] + edge_weight)
				{
					//Update Parent and EdgeWt
					SSSP[node2].Parent = node1;
					SSSP[node2].EDGwt = edge_weight;
					d_UpdatedDist[node2] = d_UpdatedDist[node1] + edge_weight;
					SSSP[node2].Update = true;
					/*printf("@@@@node: %d, parent: %d, dist: %f", node2, SSSP[node2].Parent, d_UpdatedDist[node2]);*/
					//Mark Edge to be added--node1 updated
					Edgedone[index] = 1;
					break;
				}

			}//end of for

		}//end of if insert

		//Deletion case
		//in case of deletion we don't update d_UpdatedDist
		if (allChange_cuda[index].inst == 0 && Edgedone[index] != 0)  //if deleted
		{
			Edgedone[index] = 3;
			//Check if edge exists in the tree
				//this will happen if node1 is parentof node or vice-versa
			bool iskeyedge = false;

			// printf("XXX:%d:%d \n",mye.node1, mye.node2 );

					 //Mark edge as deleted
			if (SSSP[node_1].Parent == node_2)
			{
				//printf("YYY:%d:%d \n",mye.node1, mye.node2 );
				SSSP[node_1].EDGwt = inf;
				SSSP[node_1].Update = true;
				iskeyedge = true;
				/*d_UpdatedDist[node_1] = inf;*/ //check. added recently 01-15-20
			}
			else {
				//Mark edge as deleted
				if (SSSP[node_2].Parent == node_1)
				{
					// printf("ZZZ:%d:%d \n",mye.node1, mye.node2 );
					SSSP[node_2].EDGwt = inf;
					SSSP[node_2].Update = true;
					iskeyedge = true;
					/*d_UpdatedDist[node_2] = inf;*/ //check. added recently 01-15-20
				}
			}


			//If  Key Edge Delete from key edges
		   //Set weights to -1;
			if (iskeyedge)
			{

				for (int k = 0; k < d_colStartPtr_X[node_1 + 1] - d_colStartPtr_X[node_1]; k++)
				{
					////TEPS:
					//*te = *te + 1;
					int myn = cuda_adjlist_full_X[d_colStartPtr_X[node_1] + k].col;
					if (myn == node_2)
					{
						cuda_adjlist_full_X[d_colStartPtr_X[node_1] + k].wt = -1; //set wt -1 in adj list of old sssp
						break;
					}

				}//end of for

				for (int k = 0; k < d_colStartPtr_X[node_2 + 1] - d_colStartPtr_X[node_2]; k++)
				{
					////TEPS:
					//*te = *te + 1;
					int myn = cuda_adjlist_full_X[d_colStartPtr_X[node_2] + k].col;
					if (myn == node_1)
					{
						cuda_adjlist_full_X[d_colStartPtr_X[node_2] + k].wt = -1; //set wt -1 in adj list of old sssp
						break;
					}

				}
			}//end of if


			/*else      // check. recently added 24-01-2020. The below part is required for all as we consider full graph
			{*/

			for (int k = 0; k < colStartPtr_R[node_1 + 1] - colStartPtr_R[node_1]; k++)
			{
				int myn = cuda_adjlist_full_R[colStartPtr_R[node_1] + k].col;
				if (myn == node_2)
				{
					cuda_adjlist_full_R[colStartPtr_R[node_1] + k].wt = -1;
					break;
				}

			}//end of for

			for (int k = 0; k < colStartPtr_R[node_2 + 1] - colStartPtr_R[node_2]; k++)
			{
				int myn = cuda_adjlist_full_R[colStartPtr_R[node_2] + k].col;
				if (myn == node_1)
				{
					cuda_adjlist_full_R[colStartPtr_R[node_2] + k].wt = -1;
					break;
				}

			}//end of for

		//}//end of if

		}//end of else if deleted
	}
}


__global__ void checkInsertedEdges(int numS, int* Edgedone, double* d_UpdatedDist, xEdge_cuda* allChange_cuda, RT_Vertex* SSSP, int* change_d)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < numS)
	{

		if (Edgedone[index] == 1) //Edgedone will be 1 when edge is marked to be inserted
		{

			//get the edge
			int node_1 = allChange_cuda[index].node1;
			int node_2 = allChange_cuda[index].node2;
			double edgeWeight = allChange_cuda[index].edge_wt;
			//reset it to 0
			Edgedone[index] = 0;


			int node1, node2;
			if (d_UpdatedDist[node_1] > d_UpdatedDist[node_2])
			{
				node1 = node_1;
				node2 = node_2;
			}
			else
			{
				node1 = node_2;
				node2 = node_1;
			}

			//Check if some other edge was added--mark edge to be added //check x
			if (d_UpdatedDist[node1] > d_UpdatedDist[node2] + edgeWeight)
			{
				Edgedone[index] = 1;
			}

			//Check if correct edge wt was written--mark edge to be added //check x
			if ((SSSP[node1].Parent == node2) && (SSSP[node1].EDGwt > edgeWeight))
			{
				Edgedone[index] = 1;
			}


			if (Edgedone[index] == 1)
			{
				//Update Parent and EdgeWt
				SSSP[node1].Parent = node2;
				SSSP[node1].EDGwt = edgeWeight;
				d_UpdatedDist[node1] = d_UpdatedDist[SSSP[node1].Parent] + SSSP[node1].EDGwt;
				SSSP[node2].Update = true;
				change_d[0] = 1;
			}


		}//end of if
	}
}

__global__ void updateDistance(int X_size, RT_Vertex* SSSP, double* d_UpdatedDist, double inf)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < X_size)
	{
		//do not update source node
		int px = SSSP[index].Parent;
		int flag = 0;
		if (SSSP[index].Parent == -1) { flag = 1; }


		if (flag != 1 && index == SSSP[px].Parent)
		{
			printf("DP: %d:%d %d:%d \n", index, SSSP[index].Parent, px, SSSP[px].Parent);
		}

		//For deletion case
		if (flag != 1 && SSSP[index].EDGwt == inf)
		{
			SSSP[index].Dist = inf;
			SSSP[index].Update = true;
			flag = 1;
		}

		//for insertion case
		if (flag != 1 && SSSP[index].Dist > d_UpdatedDist[index])
		{
			SSSP[index].Dist = d_UpdatedDist[index];
			/*printf("In updateDistance:  index: %d, dist:%f\n", index, SSSP[index].Dist);*/
			SSSP[index].Update = true;
		}

	}
}

__global__ void initializeUpdatedDistOldDist(double* d_UpdatedDist, double* d_OldUpdate, RT_Vertex* SSSP, int X_size)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < X_size)
	{
		d_UpdatedDist[index] = SSSP[index].Dist; //this will fill up the d_UpdatedDist values for deletion case also
		d_OldUpdate[index] = SSSP[index].Dist;
		/*printf("****Inside initializeUpdatedDistOldDist: %d edge weight: %f parent: %d dist: %f\n", index, SSSP[index].EDGwt, SSSP[index].Parent, SSSP[index].Dist);*/
	}
}


//revised function //check. recently added function. 24-01-2020
__global__ void updateNeighbors(double* d_UpdatedDist, RT_Vertex* SSSP, int X_size, /*int* d_mychange,*/ int* d_colStartPtr_X, Colwt2* cuda_adjlist_full_X, double inf, int* change_d, int its, Colwt2* cuda_adjlist_full_R, int* colStartPtr_R)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < X_size)
	{

		//If i is updated--update its neighbors as required
		if (SSSP[index].Update)
		{
			SSSP[index].Update = false;
			for (int j = 0; j < colStartPtr_R[index + 1] - colStartPtr_R[index]; j++)
			{
				int myn = cuda_adjlist_full_R[colStartPtr_R[index] + j].col;
				double mywt = cuda_adjlist_full_R[colStartPtr_R[index] + j].wt;

				if (SSSP[myn].EDGwt < mywt && SSSP[myn].Parent == index) //check if we have taken an edge with lower weight from the changeEdge set. if yes then don't update edgeweight
				{
					mywt = SSSP[myn].EDGwt;
				}
				//check if edge is deleted
				if (mywt < 0) { continue; } //if mywt = -1, that means node was deleted
				if (SSSP[index].Dist == inf)
				{
					/*printf("$$$$: %d edge weight: %f parent: %d dist: %f\n", index, SSSP[index].EDGwt, SSSP[index].Parent, SSSP[index].Dist);
					printf("$$$$myn: %d myn weight: %f parent: %d dist: %f\n", myn, mywt, SSSP[myn].Parent, SSSP[myn].Dist);
					*/
					if (myn == SSSP[index].Parent)
					{
						continue;
					}
					if (SSSP[myn].Parent == index)
					{
						d_UpdatedDist[myn] = inf;
						SSSP[myn].Dist = inf;
						SSSP[myn].Update = true;
						change_d[0] = 1;
						/*printf("&&&&index: %d edge weight: %f parent: %d dist: %f\n", index, SSSP[index].EDGwt, SSSP[index].Parent, SSSP[index].Dist);
						printf("&&&&myn: %d myn weight: %f parent: %d dist: %f\n", myn, mywt, SSSP[myn].Parent, SSSP[myn].Dist);
						*/
						continue;
					}
					else {
						if (SSSP[myn].Dist != inf)
						{
							d_UpdatedDist[index] = d_UpdatedDist[myn] + mywt;
							SSSP[index].Dist = d_UpdatedDist[myn] + mywt;
							SSSP[index].Parent = myn;
							SSSP[index].EDGwt = mywt;
							SSSP[index].Update = true;
							change_d[0] = 1;
							/*printf("++++index: %d edge weight: %f parent: %d dist: %f\n", index, SSSP[index].EDGwt, SSSP[index].Parent, SSSP[index].Dist);
							printf("++++myn: %d myn weight: %f parent: %d dist: %f\n", myn, mywt, SSSP[myn].Parent, SSSP[myn].Dist);*/

							continue;
						}
					}
				}
				if (SSSP[index].Dist != inf)
				{
					/*printf("Not inf: index: %d edge weight: %f parent: %d dist: %f\n", index, SSSP[index].EDGwt, SSSP[index].Parent, SSSP[index].Dist);
					printf("Not infmyn: %d myn weight: %f parent: %d dist: %f\n", myn, mywt, SSSP[myn].Parent, SSSP[myn].Dist);
					*/
					if (SSSP[myn].Dist == inf)
					{
						if (SSSP[index].Parent != myn)
						{
							d_UpdatedDist[myn] = d_UpdatedDist[index] + mywt;
							SSSP[myn].Dist = SSSP[index].Dist + mywt;
							SSSP[myn].EDGwt = mywt;
							SSSP[myn].Update = true;
							SSSP[myn].Parent = index;
							change_d[0] = 1;
							continue;
						}
						else {
							//don't do anything if myn is parent of index node
							continue;
						}

					}
					if (d_UpdatedDist[myn] > d_UpdatedDist[index] + mywt) //update both cases where parent of myn == index or parent of myn != index
					{
						//if (SSSP[myn].EDGwt < mywt && SSSP[myn].Parent == index) //check if we have taken an edge with lower weight from the changeEdge set. if yes then don't update edgeweight
						//{
						//	mywt = SSSP[myn].EDGwt;
						//}
						d_UpdatedDist[myn] = d_UpdatedDist[index] + mywt;
						SSSP[myn].Dist = d_UpdatedDist[index] + mywt;
						SSSP[myn].Update = true;
						SSSP[myn].Parent = index;
						change_d[0] = 1;
						/*printf("Not inf: index: %d edge weight: %f parent: %d dist: %f\n", index, SSSP[index].EDGwt, SSSP[index].Parent, SSSP[index].Dist);
						printf("Not infmyn: %d myn weight: %f parent: %d dist: %f\n", myn, mywt, SSSP[myn].Parent, SSSP[myn].Dist);
						*/
						continue;
					}
					else
					{
						if (SSSP[myn].Parent == index)
						{
							d_UpdatedDist[myn] = d_UpdatedDist[index] + mywt;
							SSSP[myn].Dist = d_UpdatedDist[index] + mywt;
							SSSP[myn].Update = true;
							/*SSSP[myn].Parent = index;*/ //Parent will remain same
							change_d[0] = 1;
							continue;
						}
						if ((d_UpdatedDist[index] > d_UpdatedDist[myn] + mywt) /*&& (SSSP[myn].Parent != index)*/)
						{
							d_UpdatedDist[index] = d_UpdatedDist[myn] + mywt;
							SSSP[index].Dist = d_UpdatedDist[myn] + mywt;
							SSSP[index].Update = true;
							SSSP[index].Parent = myn;
							change_d[0] = 1;
						}
					}
				}
			}
		}
	}
}

__global__ void checkIfDistUpdated(int X_size, double* d_OldUpdate, double* d_UpdatedDist, RT_Vertex* SSSP)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < X_size)
	{
		if (d_OldUpdate[index] != d_UpdatedDist[index])
		{
			d_OldUpdate[index] = d_UpdatedDist[index];
			SSSP[index].Update = true;
		}
		else { SSSP[index].Update = false; }
	}
}

__global__ void updateDistanceFinal(int X_size, double* d_UpdatedDist, RT_Vertex* SSSP, double inf)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < X_size)
	{
		int flag = 0;
		//do not update parent
		if (SSSP[index].Parent == -1) { flag = 1; }

		if (flag == 0)
		{
			int px = SSSP[index].Parent;
			if (px > -1)
			{
				//printf("XX %d :%d \n", i, px);  
				if (index == SSSP[px].Parent)
				{
					printf("BBP %d %d \n", index, px);
				}
			}
			if (d_UpdatedDist[index] >= inf)
			{
				SSSP[index].Dist = inf;
			}
			else
			{
				SSSP[index].Dist = d_UpdatedDist[SSSP[index].Parent] + SSSP[index].EDGwt;
				//printf("Check 2. index: %d dist: %f, parent dist:%f, edgewt: %f \n", index, SSSP[index].Dist, d_UpdatedDist[SSSP[index].Parent], SSSP[index].EDGwt); //Test 23-01-2020
			}
		}
	}
}

void edge_update(int* totalChange, int* X_size, int* SSSP_size, xEdge_cuda* allChange_cuda, Colwt2* cuda_adjlist_full_X, int* colStartPtr_X, RT_Vertex* SSSP, Colwt2* cuda_adjlist_full_R, int* colStartPtr_R, int* te, int* nodes);
void rest_update(int* X_size, Colwt2* cuda_adjlist_full_X, int* colStartPtr_X, RT_Vertex* SSSP, Colwt2* cuda_adjlist_full_R, int* colStartPtr_R, int* nodes);


/*
1st arg: original graph file name
2nd arg: input SSSP file name
3rd arg: change edges file name
4th arg: no. of nodes
5th arg: no. of edges
*/
int main(int argc, char* argv[]) {

	double startx, endx, starty, endy;
	/*double inf = std::numeric_limits<double>::infinity();*/

	/***** Preprocessing to Graph (GUI) ***********/
	int nodes, edges;
	hipError_t cudaStatus;

	/*printf("Enter number of total nodes: ");
	scanf("%d", &nodes);
	printf("Enter number of total edges: ");
	scanf("%d", &edges);
	printf("check 0");*/

	nodes = atoi(argv[4]); //when cmd line arg used
	edges = atoi(argv[5]); //when cmd line arg used


	/*** Read Full Graph ***/
	int* colStartPtr_R;
	cout << "success 1" << endl;
	cudaStatus = hipMallocManaged((void**)&colStartPtr_R, (nodes + 1) * sizeof(int)); //we take nodes +1 to store the start ptr of the first row 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		/*goto Error;*/
	}
	int total_adjmatrix_size_R = edges * 2; //e.g.= (0 1 wt1), (1 0 wt1) both are same edge, but both will be there
	Colwt2* cuda_adjlist_full_R;
	cudaStatus = hipMallocManaged(&cuda_adjlist_full_R, total_adjmatrix_size_R * sizeof(Colwt2));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		/*goto Error;*/
	}
	printf("check 1");

	//use below for direct path
	/*string file1 = "C:\\Users\\khand\\Desktop\\PhD\\CUDA test\\Test\\test 1\\fullGraph.txt";
	char* cstr1 = &file1[0];
	readin_graphU(&R, nodes, cstr1);*/

	//use below code if we use pass file name as argument
	//readin_graphU(&R, nodes, argv[1]);


	//use below code to pass the file name as relative path.
	//**keep the files in the same folder
	//string file1 = "./fullGraph.txt";
	//char* cstr1 = &file1[0];
	//readin_graphU4(colStartPtr_R, cuda_adjlist_full_R, cstr1, &nodes); //when local file used

	readin_graphU4(colStartPtr_R, cuda_adjlist_full_R, argv[1], &nodes); //when cmd line arg used

	cout << "success 2" << endl;
	/*for (int i = 0; i < nodes + 1; i++)
	{
		cout <<"*******Row start ptr: "<< colStartPtr_R[i] << endl;
			for (int y = colStartPtr_R[i]; y< colStartPtr_R[i+1];y++)
			{
				cout <<"node: "<< cuda_adjlist_full_R[y].col <<"weight: "<< cuda_adjlist_full_R[y].wt << endl;
			}
	}*/

	/*** Finished Reading Full graph **/

	/*** Read the input SSSP ***/
	int* colStartPtr_X = (int*)malloc((nodes + 1) * sizeof(int));//we take nodes +1 to store the start ptr of the first row 
	int total_adjmatrix_size_X = (nodes - 1) * 2; //maximum number of edges in SSSP tree = nodes - 1. Each edge will take 2 places in adjacent list
	Colwt2* cuda_adjlist_full_X;
	cudaStatus = hipMallocManaged(&cuda_adjlist_full_X, total_adjmatrix_size_X * sizeof(Colwt2));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at SSSP file");
		/*goto Error;*/
	}
	cout << "success 3" << endl;

	//use below code to pass the file name as relative path.
	//**keep the files in the same folder
	//string file2 = "./SSSP.txt";
	//char* cstr2 = &file2[0];
	//readin_graphU4(colStartPtr_X, cuda_adjlist_full_X, cstr2, &nodes); //when local SSSP file used

	//when cmd line arg used
	readin_graphU4(colStartPtr_X, cuda_adjlist_full_X, argv[2], &nodes); //when cmd line arg used
	cout << "success 4" << endl;

	/*for (int i = 0; i < nodes + 1; i++)
			{
				cout <<"*******Row : "<< i << endl;
					for (int y = colStartPtr_X[i]; y< colStartPtr_X[i+1];y++)
					{
						cout <<"node: "<< cuda_adjlist_full_X[y].col <<"weight: "<< cuda_adjlist_full_X[y].wt << endl;
					}
			}*/

	/*** Finished Reading input SSSP **/

	/*** Read the change file ***/
	//There will be a list for inserts and a list for delete
	vector<xEdge> allChange;
	allChange.clear();

	/*** Read set of Changed Edges ***/
   //use below for direct path
   /*string file3 = "C:\\Users\\khand\\Desktop\\PhD\\CUDA test\\Test\\test 1\\changeEdges.txt";
   char* cstr3 = &file3[0];
   readin_changes(cstr3, &allChange);*/

   //use below code if we use pass file name as argument
   /*readin_changes(argv[3], &allChange);*/

   //use below code to pass the file name as relative path.
	/*string file3 = "./changeEdges.txt";
	char* cstr3 = &file3[0];
	readin_changes(cstr3, &allChange);*/

	readin_changes(argv[3], &allChange); //when cmd line arg used

	cout << "success 5" << endl;
	//new addition
	xEdge_cuda* allChange_cuda;
	int totalChange = allChange.size();
	cudaStatus = hipMallocManaged(&allChange_cuda, totalChange * sizeof(xEdge_cuda));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at changeEdge");
		/*goto Error;*/
	}
	for (int i = 0; i < totalChange; i++)
	{
		allChange_cuda[i].node1 = allChange.at(i).theEdge.node1;
		allChange_cuda[i].node2 = allChange.at(i).theEdge.node2;
		allChange_cuda[i].edge_wt = allChange.at(i).theEdge.edge_wt;
		allChange_cuda[i].inst = allChange.at(i).inst;
	}
	/*** Finished Reading Changed Edges **/

	//Initializing  Rooted Tree
	RT_Vertex* SSSP;
	cudaStatus = hipMallocManaged(&SSSP, nodes * sizeof(RT_Vertex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at SSSP structure");
		/*goto Error;*/
	}
	int* stencil; //stencil is used for tracking which node is being affected. 
	/*hipMallocManaged(&stencil, nodes * sizeof(int));*/
	cudaStatus = hipMalloc((void**)&stencil, nodes * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at SSSP stencil");
		/*goto Error;*/
	}
	/*int* stencil_c = new int[nodes];*/
	int* stencil_c = (int*)calloc(nodes, sizeof(int));
	/*vector<SCC_vertex>SCC;*/
	int graphDirectedUndirectedIndicator = 0; // Should be 1 for SCC, 0 for not SCC. need to modify if we want SCC

	int source;
	/*printf("Enter source node: ");
	scanf("%d", &source);*/
	source = 0; //default we have taken 0 as source node

	int p;



	if (graphDirectedUndirectedIndicator == 0) {
		int src = source; //the source from which the paths are computed
		initialize << <(nodes / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (nodes, src, SSSP, stencil); //kernet call
		hipDeviceSynchronize();
		hipMemcpy(stencil_c, stencil, nodes * sizeof(int), hipMemcpyDeviceToHost);
		/*for (int i = 0; i < nodes; i++)
		{
			cout << "stencil_c" << stencil_c[i] << endl;
		}*/
		/*for (int i = 0; i < nodes; i++)
		{

			cout <<"dist"<< SSSP->Dist << endl;
			cout <<"wt"<< SSSP->EDGwt << endl;
			cout << "level"<< SSSP->Root << endl;
			cout << "marked"<< SSSP->Parent << endl;
		}*/


		//Code for create_tree:
		//Time calculation
		auto startTime = high_resolution_clock::now();
		int totalAffectedNode; //alias of numberOfAffectedNode

		int* affectedPointer;
		int* d_affectedPointer;
		cudaStatus = hipMalloc((void**)&d_affectedPointer, nodes * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed at affectedPointer");
			/*goto Error;*/
		}
		affectedPointer = (int*)calloc(nodes, sizeof(int));
		affectedPointer[0] = 1;
		hipMemcpy(d_affectedPointer, affectedPointer, nodes * sizeof(int), hipMemcpyHostToDevice);
		/*hipMallocManaged(&affectedPointer, nodes * sizeof(int));*/

		//new addition
		int* change_d = new int[1];
		int* change = new int[1];
		change[0] = 1;
		hipMalloc((void**)&change_d, 1 * sizeof(int));
		int* d_colStartPtr_X;
		hipMalloc((void**)&d_colStartPtr_X, (nodes + 1) * sizeof(int));
		hipMemcpy(d_colStartPtr_X, colStartPtr_X, (nodes + 1) * sizeof(int), hipMemcpyHostToDevice);
		while (change[0] == 1)
		{
			change[0] = 0;
			hipMemcpy(change_d, change, 1 * sizeof(int), hipMemcpyHostToDevice);
			create_tree2 << <(nodes / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>> > (cuda_adjlist_full_X, d_colStartPtr_X, SSSP, d_affectedPointer, change_d, nodes);
			hipDeviceSynchronize();
			hipMemcpy(change, change_d, 1 * sizeof(int), hipMemcpyDeviceToHost);
			/*cout << "change"<< change[0]<<endl;*/

		}

		free(affectedPointer);
		/*free(affected_nodes);*/
		hipFree(d_affectedPointer);
		hipFree(d_colStartPtr_X);

		//Time calculation
		auto stopTime = high_resolution_clock::now();
		// Time calculation
		auto duration = duration_cast<microseconds>(stopTime - startTime);
		cout << "Time taken by create-tree function: "
			<< duration.count() << " microseconds" << endl;

		//test
		/*cout << "input sssp tree" << endl;
		for (int i = 0; i < nodes; i++)
		{
			cout << "node" << i << endl;
			cout << "dist" << SSSP[i].Dist << endl;
			cout << "parent" << SSSP[i].Parent << endl;
			cout << "Edgewt" << SSSP[i].EDGwt << endl;
		}*/
		//test end
		//edge_update function
		//Update the inserted and delted edges in the tree
		int x_size = nodes;
		int SSSP_size = nodes; //considering all nodes are participating in the SSSP
		int te = 0;
		auto startTime1 = high_resolution_clock::now();
		edge_update(&totalChange, &x_size, &SSSP_size, allChange_cuda, cuda_adjlist_full_X, colStartPtr_X, SSSP, cuda_adjlist_full_R, colStartPtr_R, &te, &nodes);
		cout << "after edge_update fn" << endl;
		//Time calculation
		auto stopTime1 = high_resolution_clock::now();
		// Time calculation
		auto duration1 = duration_cast<microseconds>(stopTime1 - startTime1);
		cout << "Time taken by edge_update function: "
			<< duration1.count() << " microseconds" << endl;

		auto startTime2 = high_resolution_clock::now();
		rest_update(&x_size, cuda_adjlist_full_X, colStartPtr_X, SSSP, cuda_adjlist_full_R, colStartPtr_R, &nodes);
		cout << "after rest_update fn" << endl;
		//Time calculation
		auto stopTime2 = high_resolution_clock::now();
		// Time calculation
		auto duration2 = duration_cast<microseconds>(stopTime2 - startTime2);
		cout << "Time taken by rest_update function: "
			<< duration2.count() << " microseconds" << endl;
	}
	else
	{
		//****below code needs modification
		/*SCC.clear();
		readin_SCC(argv[2], &SCC);
		update_SCC(&X, &SCC, &allChange);
		print_network(X);*/
	}
	//Test code start
	cout << "SSSP" << endl;
	for (int i = 0; i < nodes; i++)
	{
		cout << "*******" << endl;
		cout << "node" << i << endl << "dist" << SSSP[i].Dist << endl << "parent" << SSSP[i].Parent << endl;
	}
	cout << "*******success*******" << endl;
	
	//Test code end


	hipFree(colStartPtr_R);
	hipFree(cuda_adjlist_full_R);
	hipFree(colStartPtr_X);
	hipFree(cuda_adjlist_full_X);
	hipFree(allChange_cuda);
	hipFree(SSSP);
	hipFree(stencil);
Error:
	hipFree(colStartPtr_R);
	return 0;
}



void edge_update(int* totalChange, int* X_size, int* SSSP_size, xEdge_cuda* allChange_cuda, Colwt2* cuda_adjlist_full_X, int* colStartPtr_X, RT_Vertex* SSSP, Colwt2* cuda_adjlist_full_R, int* colStartPtr_R, int* te, int* nodes)
{
	double inf = std::numeric_limits<double>::infinity();
	/*int* Edgedone;*/
	double* UpdatedDist;

	int iter = 0;

	//Mark how the edge is processed
	int* Edgedone;
	hipMalloc((void**)&Edgedone, (*totalChange) * sizeof(int));
	//initialize Edgedone array with -1
	initializeEdgedone << <((*totalChange) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (Edgedone, *totalChange);
	hipDeviceSynchronize();

	/*thrust::device_vector<int> Edgedone_ptr(*totalChange);
	thrust::fill(Edgedone_ptr.begin(), Edgedone_ptr.end(), -1);
	int* Edgedone = thrust::raw_pointer_cast(Edgedone_ptr);*/

	//Store the updated distance value
	UpdatedDist = (double*)calloc(*X_size, sizeof(double));
	double* d_UpdatedDist;
	hipMalloc((void**)&d_UpdatedDist, (*X_size) * sizeof(double));
	hipMemcpy(d_UpdatedDist, UpdatedDist, (*X_size) * sizeof(double), hipMemcpyHostToDevice);

	//Initialize with current distance for each node
	initializeUpdatedDist << <((*X_size) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (d_UpdatedDist, SSSP, *X_size);
	hipDeviceSynchronize();
	/*	hipMemcpy(UpdatedDist, d_UpdatedDist, (*X_size) * sizeof(double), hipMemcpyDeviceToHost);*/ //not required


	int numS = *totalChange;
	int* d_colStartPtr_X;
	hipMalloc((void**)&d_colStartPtr_X, (*nodes + 1) * sizeof(int));
	hipMemcpy(d_colStartPtr_X, colStartPtr_X, (*nodes + 1) * sizeof(int), hipMemcpyHostToDevice);

	insertDeleteEdge << < (numS / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (allChange_cuda, Edgedone, SSSP, numS, *X_size, d_colStartPtr_X, cuda_adjlist_full_X, d_UpdatedDist, inf, cuda_adjlist_full_R, colStartPtr_R);
	hipDeviceSynchronize();


	/*int* Edgedone_c = new int[*totalChange];
	hipMemcpy(Edgedone_c, Edgedone, *totalChange * sizeof(int), hipMemcpyDeviceToHost); *///not req.
	/*hipMemcpy(UpdatedDist, d_UpdatedDist, (*X_size) * sizeof(double), hipMemcpyDeviceToHost); *///not req.


	//Go over the inserted edges to see if they need to be changed
	int* change_d = new int[1];
	int* change = new int[1];
	change[0] = 1;
	hipMalloc((void**)&change_d, 1 * sizeof(int));
	/*hipMemcpy(change_d, change, 1 * sizeof(int), hipMemcpyHostToDevice);*/ //recent change
	while (change[0] == 1)
	{
		change[0] = 0;
		hipMemcpy(change_d, change, 1 * sizeof(int), hipMemcpyHostToDevice);
		checkInsertedEdges << < (numS / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (numS, Edgedone, d_UpdatedDist, allChange_cuda, SSSP, change_d);
		hipDeviceSynchronize();
		hipMemcpy(change, change_d, 1 * sizeof(int), hipMemcpyDeviceToHost);
		/*cout << "change"<< change[0]<<endl;*/

	}

	//Update the distances
	 //Initialize with current distance for each node
	updateDistance << < (numS / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (*X_size, SSSP, d_UpdatedDist, inf);
	hipDeviceSynchronize();


	hipFree(change_d);
	hipFree(d_UpdatedDist);
	hipFree(d_colStartPtr_X);
	free(UpdatedDist);
	return;
}


void rest_update(int* X_size, Colwt2* cuda_adjlist_full_X, int* colStartPtr_X, RT_Vertex* SSSP, Colwt2* cuda_adjlist_full_R, int* colStartPtr_R, int* nodes)
{
	double inf = std::numeric_limits<double>::infinity();


	int its = 0; //number of iterations

	int* change_d = new int[1];
	int* change = new int[1]; //marking whether the connections changed in the iteration
	change[0] = 1;
	hipMalloc((void**)&change_d, 1 * sizeof(int));
	/*hipMemcpy(change_d, change, 1 * sizeof(int), hipMemcpyHostToDevice);*/ //recent change

	double* UpdatedDist;
	//Store the updated distance value
	UpdatedDist = (double*)calloc(*X_size, sizeof(double));
	double* d_UpdatedDist;
	hipMalloc((void**)&d_UpdatedDist, (*X_size) * sizeof(double));
	hipMemcpy(d_UpdatedDist, UpdatedDist, (*X_size) * sizeof(double), hipMemcpyHostToDevice);


	double* OldUpdate;
	//Store the old updated distance value
	OldUpdate = (double*)calloc(*X_size, sizeof(double));
	double* d_OldUpdate;
	hipMalloc((void**)&d_OldUpdate, (*X_size) * sizeof(double));
	hipMemcpy(d_OldUpdate, OldUpdate, (*X_size) * sizeof(double), hipMemcpyHostToDevice);


	//int* mychange;
	////Store the old updated distance value
	//mychange = (int*)calloc(*X_size, sizeof(int));
	//int* d_mychange;
	//hipMalloc((void**)&d_mychange, (*X_size) * sizeof(int));
	//hipMemcpy(d_mychange, mychange, (*X_size) * sizeof(int), hipMemcpyHostToDevice);

	int* d_colStartPtr_X;
	hipMalloc((void**)&d_colStartPtr_X, (*nodes + 1) * sizeof(int));
	hipMemcpy(d_colStartPtr_X, colStartPtr_X, (*nodes + 1) * sizeof(int), hipMemcpyHostToDevice);


	//Initialize with current distance for each node
	initializeUpdatedDistOldDist << <((*X_size) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (d_UpdatedDist, d_OldUpdate, SSSP, *X_size);
	hipDeviceSynchronize();


	int iter = 0;
	while (change[0] == 1 && its < 70)
	{
		printf("Iteration:%d \n", its);

		change[0] = 0;
		hipMemcpy(change_d, change, 1 * sizeof(int), hipMemcpyHostToDevice);
		updateNeighbors << <((*X_size) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (d_UpdatedDist, SSSP, *X_size, /*d_mychange,*/ d_colStartPtr_X, cuda_adjlist_full_X, inf, change_d, its, cuda_adjlist_full_R, colStartPtr_R);
		hipDeviceSynchronize();
		hipMemcpy(change, change_d, 1 * sizeof(int), hipMemcpyDeviceToHost);

		//Test code start
		/*hipMemcpy(UpdatedDist, d_UpdatedDist, (*X_size) * sizeof(double), hipMemcpyDeviceToHost);
		for (int i = 0; i < *X_size; i++)
		{
			cout << "UpdatedDist: " << UpdatedDist[i] << endl;
		}*/
		//Test code end


	//Check if distance was updated
		checkIfDistUpdated << <((*X_size) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (*X_size, d_OldUpdate, d_UpdatedDist, SSSP);
		hipDeviceSynchronize();
		its++;
	}//end of while
	printf("Total Iterations to Converge %d \n", its);

	//Update the distances
	//Initialize with current distance for each node
	updateDistanceFinal << <((*X_size) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (*X_size, d_UpdatedDist, SSSP, inf);
	hipDeviceSynchronize();


	free(UpdatedDist);
	free(OldUpdate);
	/*free(mychange);*/
	hipFree(change_d);
	hipFree(d_UpdatedDist);
	hipFree(d_OldUpdate);
	/*hipFree(d_mychange);*/
	hipFree(d_colStartPtr_X);

	return;
}

